
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftPlus.cu"
#else

#include "../common.h"

void THNN_(SoftPlus_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal beta_,
           accreal threshold_)
{
  real beta = ScalarConvert<accreal, real>::to(beta_);
  real threshold = ScalarConvert<accreal, real>::to(threshold_);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2<real, real>(state, output, input, softPlusupdateOutput_functor<real>(threshold, beta));
}

void THNN_(SoftPlus_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           accreal beta_,
           accreal threshold_)
{
  real beta = ScalarConvert<accreal, real>::to(beta_);
  real threshold = ScalarConvert<accreal, real>::to(threshold_);
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 4, input, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3<real, real, real>(state, gradInput, output, gradOutput, softPlusupdateGradInput_functor<real>(threshold, beta));
}

#endif
